#include "hip/hip_runtime.h"
// please be noted that This programm will not work without nvidia cuda toolKit, or with google colab
// Odd-Even algorithm but in parallel using cuda instructions
// the function is in the device (GPU), but the main is in the host (CPU) 
%%cu
#include<stdio.h>
#include<hip/hip_runtime.h>
#define swap(A,B) {int temp=A;A=B;B=temp;}

__global__ void OddEvenSort(int *d_arr , int size, int step)
{
  
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(idx % 2 == step && idx < size - 1)
    if(d_arr[idx] > d_arr[idx + 1]){
      swap(d_arr[idx],d_arr[idx + 1]);
    }
  __syncthreads();
}


int main(void)
{
  int blocks  = 1024;
  int threads = 1024;
  int arr[30];
  int *d_arr;
  int n       = sizeof(arr) / sizeof(arr[0]);
  int size    = sizeof(int) * n;
  int a;


  printf("The original array  : {");
 
  // Random variable assigned for the array  
  for (int i = 0; i < n - 2; i++) 
  {
    arr[i] = rand()%29; 
    printf(" %d ,", arr[i]);
  }
  arr[29] = rand()%29; 
  printf(" %d ", arr[29]);
  printf("}\n");


  // allocate array in device
  hipMalloc(&d_arr , size);

  // copy host copy to device copy array
  hipMemcpy(d_arr , arr , size , hipMemcpyHostToDevice);


  if( n % 2 == 0)
    a = n / 2;
    else
      a = n / 2 + 1;

  // calling the kernel
  for (int i = 0; i < a; i++)
  {
    
    // odd step
    OddEvenSort <<< blocks , threads >>> (d_arr, n, 1);
   

    // even step
    OddEvenSort <<< blocks , threads >>> (d_arr, n, 0);
    
  }

  hipDeviceSynchronize();

  // copy results back to host array
  hipMemcpy(arr , d_arr , size , hipMemcpyDeviceToHost);

  // free
  hipFree(d_arr);

  // print results
  printf("array after sorting : { ");
  for (int i = 0 ; i < n - 2 ; i++)
    printf("%d , ", arr[i]);

  printf("%d }", arr[28]);

return 0;
}